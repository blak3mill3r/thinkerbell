/*
 * Class DeviceResource
 * simple encapsulation of page-locked host
 * memory and device memory and transfers
 * between them
 * automatically allocates/deallocates with the scope of an instance (RAII)
 * never implicitly performs host/device transfers
 */

#include <iostream>
#include "device_resource.h"

//FIXME make the throws more useful
DeviceResource::DeviceResource( uint bytes )
{
  hipError_t result;

  // allocate (page-locked) host memory
  result = hipHostAlloc( (void**) &m_host_pointer, bytes, hipHostMallocDefault );
  if(result != hipSuccess) { throw 1; }
  //m_host_pointer = malloc( bytes );

  // allocate device memory
  result = hipMalloc((void**) &m_device_pointer, bytes );
  if(result != hipSuccess) { throw 2; }
}

DeviceResource::~DeviceResource()
{
  hipError_t result;
  result = hipFree( m_device_pointer );
  if(result != hipSuccess) { throw 3; }
  result = hipHostFree( m_host_pointer );
  if(result != hipSuccess) { throw 4; }
}

void DeviceResource::host_to_device()
{
  hipError_t result;
  result = hipMemcpy(m_device_pointer,
                      m_host_pointer,
                      m_bytes,
                      hipMemcpyHostToDevice);
  if(result != hipSuccess) { std::cout << "Badness, it's not hipSuccess it is instead " << result << "\n"; throw 5; }
}

void DeviceResource::device_to_host()
{
  hipError_t result;
  result = hipMemcpy(m_host_pointer,
                      m_device_pointer,
                      m_bytes,
                      hipMemcpyDeviceToHost);
  if(result != hipSuccess) { throw 6; }
}


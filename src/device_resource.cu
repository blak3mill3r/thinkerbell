/*
 * Class DeviceResource
 * simple encapsulation of page-locked host
 * memory and device memory and transfers
 * between them
 * automatically allocates/deallocates with the scope of an instance (RAII)
 * never implicitly performs host/device transfers
 */

#include "device_resource.h"

//cutilSafeCall(
//);
DeviceResource::DeviceResource( uint bytes )
{
  // allocate (page-locked) host memory
  hipHostAlloc( (void**) &m_host_pointer, bytes, hipHostMallocDefault );
  // allocate device memory
  hipMalloc((void**) &m_device_pointer, bytes );
}

DeviceResource::~DeviceResource()
{
  hipFree( m_device_pointer );
  hipHostFree( m_host_pointer );
}

void DeviceResource::host_to_device()
{
  hipMemcpy(m_device_pointer,
             m_host_pointer,
             m_bytes,
             hipMemcpyHostToDevice);
}

void DeviceResource::device_to_host()
{
  hipMemcpy(m_host_pointer,
             m_device_pointer,
             m_bytes,
             hipMemcpyDeviceToHost);
}

